// This program is a proof of concept for threading the reconstruction
// to the GPUs as separate short-stack reconstructions and then stitching
// after all threads have been completed.

// If the code behaves as it should you should observe some stuff being printed
// as the code initializes (this will likely come out of order since the code is)
// threaded and then the numbers 0.0-100.0 printed out in increments of 1.0.  These
// should be *in order* although you may observe that the last values are 0.0 when it should
// be 100.  This is not reflective of the method, but just a blocking issue that's too minor
// to bother fixing.

// To make this run on your machine it is likely that paths will need to be modified.

// Compile on linux with:
//      nvcc main.cu -std=c++11 -o multi_gpu_test -lm

// Compile in visual studio by adding main.cu to a cuda project and making sure that the c++
// math library is being linked. Code required c++11 for threading.

#include <string.h>
#include <math.h>

#include <cstdio>
#include <iostream>
#include <fstream>
#include <vector>
#include <thread>

#include <hip/hip_runtime.h>

__global__ void recon_kernel(float * d_array,float start, float slice_thickness){
    d_array[threadIdx.x] = threadIdx.x*slice_thickness + start;
}

void dummy_recon_cpu(float start, float end, float slice_thickness,std::string output_filepath){

    int n_slices = (end - start)/slice_thickness + 1;
    float * array = new float[n_slices];

    printf("Block start: %.2f\n",start);
    printf("Block end: %.2f\n"  ,end);
    printf("Block slices: %d\n" ,n_slices);
    
    for (int i=0;i<n_slices;i++){
        array[i] = i*slice_thickness + start;
        //printf("%.02f\n",array[i]);
    }

    std::cout << output_filepath << std::endl;
    std::ofstream outfile(output_filepath,std::ios::binary | std::ios::out);
    outfile.write((char *)&array[0],n_slices*sizeof(float));
    outfile.close();
    
    delete[] array;
}

void dummy_recon_gpu(float start, float end, float slice_thickness,std::string output_filepath,int device_idx){
    
    printf("Block GPU: %d\n",device_idx);
    //printf("Block start: %.2f\n",start);
    //printf("Block end: %.2f\n"  ,end);
    //printf("Block slices: %d\n" ,n_slices);

    hipSetDevice(device_idx);

    int n_slices = (end - start)/slice_thickness + 1;
    float * h_array = new float[n_slices];
    float * d_array;
    hipMalloc(&d_array,n_slices*sizeof(float));

    recon_kernel<<<1,n_slices>>>(d_array,start,slice_thickness);
    hipMemcpy(h_array,d_array,n_slices*sizeof(float),hipMemcpyDeviceToHost);
    
    std::cout << output_filepath << std::endl;
    std::ofstream outfile(output_filepath,std::ios::binary | std::ios::out);
    outfile.write((char *)&h_array[0],n_slices*sizeof(float));
    outfile.close();

    hipFree(d_array);
    delete[] h_array;

}

int main(int argc, char ** argv){
    // ********************
    // Initial config stuff
    // ********************    
    std::string output_filepath = "/home/john/Code/multigpu_test/test.bin";
    
    int n_devices;
    hipGetDeviceCount(&n_devices);
    std::cout << "Cuda Devices Found: " << n_devices << std::endl;

    float start = 0.0f;
    float end   = 100.0f;
    float slice_thickness = 1.0f;
    int n_slices = (end-start)/slice_thickness + 1;

    // ********************    
    // Dispatch threads to do recon in blocks
    // ********************    
    // configure block metadata
    float block_n_slices = ceil(n_slices / n_devices);
    float block_start = start;
    float block_end = (block_n_slices-1)*slice_thickness + block_start;
    
    std::vector<std::thread> threads;    
    for (int i=0;i<n_devices;i++){
        std::string filename = "/tmp/recon_";

        printf("===========================\n");
        printf("Block %d\n",i);
        printf("===========================\n");

        //threads.emplace_back(dummy_recon_cpu,block_start,block_end,slice_thickness,filename + std::to_string(i) + ".bin");
        threads.emplace_back(dummy_recon_gpu,block_start,block_end,slice_thickness,filename + std::to_string(i) + ".bin",i);        
        block_start = block_end + slice_thickness;
        block_end   = std::min((block_n_slices-1)*slice_thickness + block_start,end);        
    }

    // Wait for threads to finish
    for (int i=0;i<n_devices;i++){
        threads[i].join();
    }

    std::cout << "ALL THREADS COMPLETE" << std::endl;
    std::cout << "Showing reassembled slice locations" << std::endl;
    
    // ********************
    // Reassemble all files and move to a final location
    // ********************    
    std::ofstream final_recon(output_filepath.c_str(),std::ios::binary | std::ios::out);
    for (int i=0;i<n_devices;i++){
        
        std::string filename = "/tmp/recon_" + std::to_string(i) + ".bin";
        std::ifstream tmp(filename,std::ios::binary | std::ios::in);
        if (!tmp.good())
            std::cout << "Could not find file: " << filename << std::endl;
        final_recon << tmp.rdbuf();
        tmp.close();
    }
    final_recon.close();

    // ********************    
    // Read the final file and print
    // ********************    
    float *tmp_array = new float[n_slices];
    std::ifstream final_recon_in(output_filepath.c_str(),std::ios::binary);
    final_recon_in.read((char*)&tmp_array[0],n_slices*sizeof(float));
    for (int i=0; i<n_slices; i++){
        printf("%.02f\n",tmp_array[i]);
    }

    delete[] tmp_array;

    return 0;    
}


    
